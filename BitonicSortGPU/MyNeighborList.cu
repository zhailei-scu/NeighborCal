#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>
#include "MyNeighborList.h"

#define BLOCKSIZE 512

__global__ void Kernel_MyNeighborListCal(int NClusters, double** Dev_ClustersPosXYZ, int* SortedIndex, int* Dev_NNearestNeighbor) {
	int tid = threadIdx.y*blockDim.x + threadIdx.x;
	int bid = blockIdx.y*gridDim.x + blockIdx.x;
	int cid = bid * BLOCKSIZE + tid;
	int LeftBound;
	int RightBound;
	int sortedID;
	double Pos_X;
	double Pos_Y;
	double Pos_Z;
	int relativeIC;
	double distance;
	double minDistance;
	double distanceX;
	double distanceY;
	double distanceZ;
	int NNID;
	bool exitFlag;
	int NRemind;
	int MapedIdex;
	__shared__ double Shared_XYZ[BLOCKSIZE][3];
	__shared__ int Shared_SortedID[BLOCKSIZE];
	__shared__ int NExitedThreadsRight;
	__shared__ int NExitedThreadsLeft;

	minDistance = 1.E32;

	/*Right Hand Searching*/
	exitFlag = false;
	LeftBound = bid * BLOCKSIZE;
	if (LeftBound < 0) LeftBound = 0;
	RightBound = (bid + 1)*BLOCKSIZE;
	if (RightBound > NClusters) RightBound = NClusters;

	NRemind = RightBound - LeftBound;

	NExitedThreadsRight = 0;

	if (cid < NClusters) {

		MapedIdex = SortedIndex[cid];

		Pos_X = Dev_ClustersPosXYZ[MapedIdex][0];
		Pos_Y = Dev_ClustersPosXYZ[MapedIdex][1];
		Pos_Z = Dev_ClustersPosXYZ[MapedIdex][2];

	}

	while (LeftBound < RightBound) {

		if (NExitedThreadsRight >= NRemind) {
			break;
		}

		if ((LeftBound + tid) < NClusters) {

			sortedID = SortedIndex[LeftBound + tid];

			Shared_SortedID[tid] = sortedID;

			Shared_XYZ[tid][0] = Dev_ClustersPosXYZ[sortedID][0];

			Shared_XYZ[tid][1] = Dev_ClustersPosXYZ[sortedID][1];
			Shared_XYZ[tid][2] = Dev_ClustersPosXYZ[sortedID][2];
		}

		__syncthreads();


		if (cid < NClusters) {

			if (false == exitFlag) {
				for (int IC = LeftBound; IC < RightBound; IC++) {
					if (IC != cid) {

						relativeIC = IC - LeftBound;

						distanceX = Shared_XYZ[relativeIC][0] - Pos_X;
						distanceY = Shared_XYZ[relativeIC][1] - Pos_Y;
						distanceZ = Shared_XYZ[relativeIC][2] - Pos_Z;

						distanceX = distanceX * distanceX;
						distanceY = distanceY * distanceY;
						distanceZ = distanceZ * distanceZ;

						distance = distanceX + distanceY + distanceZ;

						if (minDistance > distance) {
							minDistance = distance;
							NNID = Shared_SortedID[relativeIC];
						}

						if (distanceX > minDistance) {
							exitFlag = true;
							atomicAdd_block(&NExitedThreadsRight, 1);
							break;
						}

					}

				}
			}

		}

		__syncthreads();

		LeftBound = LeftBound + BLOCKSIZE;
		RightBound = RightBound + BLOCKSIZE;
		if (RightBound > NClusters) RightBound = NClusters;
	}

	/*Left Hand Searching*/
	exitFlag = false;
	LeftBound = (bid - 1) * BLOCKSIZE;
	if (LeftBound < 0) LeftBound = 0;
	RightBound = bid * BLOCKSIZE;
	if (RightBound > NClusters) RightBound = NClusters;

	NExitedThreadsLeft = 0;

	while (LeftBound < RightBound) {

		if (NExitedThreadsLeft >= NRemind) {
			break;
		}


		if ((LeftBound + tid) < NClusters) {

			sortedID = SortedIndex[LeftBound + tid];

			Shared_SortedID[tid] = sortedID;

			Shared_XYZ[tid][0] = Dev_ClustersPosXYZ[sortedID][0];

			Shared_XYZ[tid][1] = Dev_ClustersPosXYZ[sortedID][1];
			Shared_XYZ[tid][2] = Dev_ClustersPosXYZ[sortedID][2];

		}

		__syncthreads();

		if (cid < NClusters) {

			if (false == exitFlag) {
				for (int IC = RightBound -1; IC >= LeftBound; IC--) {
					if (IC != cid) {

						relativeIC = IC - LeftBound;

						distanceX = Shared_XYZ[relativeIC][0] - Pos_X;
						distanceY = Shared_XYZ[relativeIC][1] - Pos_Y;
						distanceZ = Shared_XYZ[relativeIC][2] - Pos_Z;

						distanceX = distanceX * distanceX;
						distanceY = distanceY * distanceY;
						distanceZ = distanceZ * distanceZ;

						distance = distanceX + distanceY + distanceZ;

						if (minDistance > distance) {
							minDistance = distance;
							NNID = Shared_SortedID[relativeIC];
						}

						if (distanceX > minDistance) {
							exitFlag = true;
							atomicAdd_block(&NExitedThreadsLeft, 1);
							break;
						}

					}

				}
			}

		}

		__syncthreads();

		RightBound = LeftBound;
		LeftBound = LeftBound - BLOCKSIZE;
		if(LeftBound < 0) LeftBound = 0;
	}


	if(cid < NClusters){
		Dev_NNearestNeighbor[MapedIdex] = NNID;
	}

}



__global__ void Kernel_NormalCalcNeighborList(int NClusters, double** Dev_ClustersPosXYZ, int* Dev_NNearestNeighbor) {
	int tid = threadIdx.y*blockDim.x + threadIdx.x;
	int bid = blockIdx.y*gridDim.x + blockIdx.x;
	int cid = bid * BLOCKSIZE + tid;
	int LeftBound;
	int RightBound;
	double Pos_X;
	double Pos_Y;
	double Pos_Z;
	int relativeIC;
	double distance;
	double minDistance;
	int NNID;
	__shared__ double Shared_XYZ[BLOCKSIZE][3];

	minDistance = 1.E32;

	/*Right Hand Searching*/
	LeftBound = 0;
	RightBound = BLOCKSIZE;
	//RightBound = RightBound < NClusters ? RightBound : NClusters;

	if (RightBound > NClusters) RightBound = NClusters;

	if (cid < NClusters) {
		Pos_X = Dev_ClustersPosXYZ[cid][0];
		Pos_Y = Dev_ClustersPosXYZ[cid][1];
		Pos_Z = Dev_ClustersPosXYZ[cid][2];
	}

	while (LeftBound < RightBound) {

			if ((LeftBound + tid)< NClusters) {
				Shared_XYZ[tid][0] = Dev_ClustersPosXYZ[LeftBound + tid][0];
				Shared_XYZ[tid][1] = Dev_ClustersPosXYZ[LeftBound + tid][1];
				Shared_XYZ[tid][2] = Dev_ClustersPosXYZ[LeftBound + tid][2];
			}

			__syncthreads();

			if (cid < NClusters) {

				for (int IC = LeftBound; IC < RightBound; IC++) {
					if (IC != cid) {

						relativeIC = IC - LeftBound;

						distance = (Shared_XYZ[relativeIC][0] - Pos_X)*(Shared_XYZ[relativeIC][0] - Pos_X) +
							(Shared_XYZ[relativeIC][1] - Pos_Y)*(Shared_XYZ[relativeIC][1] - Pos_Y) +
							(Shared_XYZ[relativeIC][2] - Pos_Z)*(Shared_XYZ[relativeIC][2] - Pos_Z);

						if (distance < minDistance) {
							NNID = IC;

							minDistance = distance;
						}

					}

				}

			}

			__syncthreads();

			LeftBound = LeftBound + BLOCKSIZE;
			RightBound = RightBound + BLOCKSIZE;
			//RightBound = RightBound < NClusters ? RightBound : NClusters;
			if (RightBound > NClusters) RightBound = NClusters;

	}

	if (cid < NClusters) {
		Dev_NNearestNeighbor[cid] = NNID;
	}

}



void My_NeighborListCal_RadixSort(int NClusters, double* ToSortDev_ClustersPosX, double** Dev_ClustersPosXYZ, int* SortedIndex, int* Dev_NNearestNeighbor,int* Host_NNearestNeighbor,float &timerMyMethod) {
	dim3 threads;
	dim3 blocks;
	int NB;
	hipError_t err;
	int noone;

	hipEvent_t StartEvent;
	hipEvent_t StopEvent;

	hipEventCreate(&StartEvent);
	hipEventCreate(&StopEvent);

	hipEventRecord(StartEvent, 0);

	thrust::device_ptr<double> Device_thrust_Key(ToSortDev_ClustersPosX);
	thrust::device_ptr<int> Device_thrust_Value(SortedIndex);

	NB = (NClusters - 1) / BLOCKSIZE + 1;

	blocks = dim3(NB,1,1);
	threads = dim3(BLOCKSIZE, 1, 1);
	thrust::sort_by_key(Device_thrust_Key, Device_thrust_Key + NClusters, Device_thrust_Value);

	Kernel_MyNeighborListCal << < blocks, threads >> > (NClusters, Dev_ClustersPosXYZ, SortedIndex,Dev_NNearestNeighbor);

	hipDeviceSynchronize();

	hipEventRecord(StopEvent,0);

	hipEventSynchronize(StopEvent);

	hipEventElapsedTime(&timerMyMethod,StartEvent,StopEvent);

	hipMemcpy(Host_NNearestNeighbor, Dev_NNearestNeighbor,NClusters*sizeof(int),hipMemcpyDeviceToHost);

	hipEventDestroy(StartEvent);
	hipEventDestroy(StopEvent);

}

void My_NeighborListCal_ArbitrayBitonicSort(int NClusters, double* ToSortDev_ClustersPosX, double** Dev_ClustersPosXYZ, int* SortedIndex, int* Dev_NNearestNeighbor, int* Host_NNearestNeighbor, float &timerMyMethod) {
	dim3 threads;
	dim3 blocks;
	int NB;
	hipError_t err;
	int noone;

	hipEvent_t StartEvent;
	hipEvent_t StopEvent;

	hipEventCreate(&StartEvent);
	hipEventCreate(&StopEvent);

	hipEventRecord(StartEvent, 0);

	thrust::device_ptr<double> Device_thrust_Key(ToSortDev_ClustersPosX);
	thrust::device_ptr<int> Device_thrust_Value(SortedIndex);

	NB = (NClusters - 1) / BLOCKSIZE + 1;

	blocks = dim3(NB, 1, 1);
	threads = dim3(BLOCKSIZE, 1, 1);
	thrust::sort_by_key(Device_thrust_Key, Device_thrust_Key + NClusters, Device_thrust_Value);

	Kernel_MyNeighborListCal << < blocks, threads >> > (NClusters, Dev_ClustersPosXYZ, SortedIndex, Dev_NNearestNeighbor);

	hipDeviceSynchronize();

	hipEventRecord(StopEvent, 0);

	hipEventSynchronize(StopEvent);

	hipEventElapsedTime(&timerMyMethod, StartEvent, StopEvent);

	hipMemcpy(Host_NNearestNeighbor, Dev_NNearestNeighbor, NClusters * sizeof(int), hipMemcpyDeviceToHost);

	hipEventDestroy(StartEvent);
	hipEventDestroy(StopEvent);

}


void Common_NeighborListCal(int NClusters, double** Dev_ClustersPosXYZ, int* Dev_NNearestNeighbor, int* Host_NNearestNeighbor,float &timerCommonGPU) {
	dim3 threads;
	dim3 blocks;
	int NB;

	hipEvent_t StartEvent;
	hipEvent_t StopEvent;

	hipEventCreate(&StartEvent);
	hipEventCreate(&StopEvent);

	hipEventRecord(StartEvent,0);

	NB = (NClusters - 1) / BLOCKSIZE + 1;

	blocks = dim3(NB, 1, 1);
	threads = dim3(BLOCKSIZE, 1, 1);


	Kernel_NormalCalcNeighborList << < blocks, threads >> > (NClusters, Dev_ClustersPosXYZ, Dev_NNearestNeighbor);

	hipDeviceSynchronize();

	hipEventRecord(StopEvent, 0);

	hipEventSynchronize(StopEvent);

	hipEventElapsedTime(&timerCommonGPU,StartEvent,StopEvent);

	hipMemcpy(Host_NNearestNeighbor, Dev_NNearestNeighbor, NClusters * sizeof(int), hipMemcpyDeviceToHost);
}


void Common_NeighborListCal_CPU(int NClusters, double** Host_ClustersPosXYZ, int* Host_NNearestNeighbor) {

	double minDist;
	double Distance;
	for (int i = 0; i < NClusters; i++) {

		minDist = 1.E16;

		for (int j = 0; j < NClusters; j++) {
			if (i != j) {
				Distance = (Host_ClustersPosXYZ[i][0] - Host_ClustersPosXYZ[j][0])*(Host_ClustersPosXYZ[i][0] - Host_ClustersPosXYZ[j][0]) + \
					(Host_ClustersPosXYZ[i][1] - Host_ClustersPosXYZ[j][1])*(Host_ClustersPosXYZ[i][1] - Host_ClustersPosXYZ[j][1]) + \
					(Host_ClustersPosXYZ[i][2] - Host_ClustersPosXYZ[j][2])*(Host_ClustersPosXYZ[i][2] - Host_ClustersPosXYZ[j][2]);

				if (Distance < minDist) {
					minDist = Distance;
					Host_NNearestNeighbor[i] = j;
				}

			}
		}
	}

}